#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hemi.h"
#include "common.h"
#include <omp.h>

// assert() is only supported // for devices of compute capability 2.0 and higher 
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200) 
#undef  assert 
#define assert(arg) 
#endif

int get_cuda_device_count()
{
   int count = 1;
   hipGetDeviceCount(&count);
   return count;
}

__constant__ double c_tau;
__constant__ double c_h;
__constant__ double c_tau_to_current_time_level;
__constant__ double c_lb;
__constant__ double c_rb;
__constant__ double c_ub;
__constant__ double c_bb;
__constant__ double c_tau_b;
__constant__ double c_pi_half;
__constant__ int c_x_length;
__constant__ int c_n;

__global__ void get_square_coord(double* first1, double* second1, double* third1,
	double* first2, double* second2, double* third2)
{
	for (int opt = blockIdx.x * blockDim.x + threadIdx.x; opt < c_n; opt += blockDim.x * gridDim.x)
	{
		int i = opt % c_x_length + 1;
		int j = opt / c_x_length + 1;
		double x, y;

		// A
		x = (c_h*(i - 1) + c_h*i) / 2.;
		y = (c_h*(j - 1) + c_h*j) / 2.;
		first1[2 * opt] = first2[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		first1[2 * opt + 1] = first2[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));

		// B
		x = (c_h*(i + 1) + c_h*i) / 2.;
		//	y = (c_h*(j - 1) + c_h*j) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		second1[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		second1[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));

		// C
		//x = (a_x[i + 1] + a_x[i]) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		y = (c_h*(j + 1) + c_h*j) / 2.;
		third1[2 * opt] = third2[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		third1[2 * opt + 1] = third2[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));

		// D 
		x = (c_h*(i - 1) + c_h*i) / 2.;
		//y = (a_y[j + 1] + a_y[j]) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		second2[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		second2[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));
	}
}

__global__ void get_square_coord_first1_2(double *result)
{
	for (int opt = blockIdx.x * blockDim.x + threadIdx.x; opt < c_n; opt += blockDim.x * gridDim.x)
	{
		int i = opt % c_x_length + 1;
		int j = opt / c_x_length + 1;
		double x, y;

		// A
		x = (c_h*(i - 1) + c_h*i) / 2.;
		y = (c_h*(j - 1) + c_h*j) / 2.;
		result[2 * opt] =  x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		result[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));
	}
}

__global__ void get_square_coord_second1(double *result)
{
	for (int opt = blockIdx.x * blockDim.x + threadIdx.x; opt < c_n; opt += blockDim.x * gridDim.x)
	{
		int i = opt % c_x_length + 1;
		int j = opt / c_x_length + 1;

		double x, y;
// B
		x = (c_h*(i + 1) + c_h*i) / 2.;
		result[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		result[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));
		
	}
}

__global__ void get_square_coord_third1_2(double *result)
{
	for (int opt = blockIdx.x * blockDim.x + threadIdx.x; opt < c_n; opt += blockDim.x * gridDim.x)
	{
		int i = opt % c_x_length + 1;
		int j = opt / c_x_length + 1;

		double x, y;
// C
		//x = (a_x[i + 1] + a_x[i]) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		y = (c_h*(j + 1) + c_h*j) / 2.;
		result[2 * opt]  = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		result[2 * opt + 1]  = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));
		
	}
}

__global__ void get_square_coord_second2(double *result)
{
	for (int opt = blockIdx.x * blockDim.x + threadIdx.x; opt < c_n; opt += blockDim.x * gridDim.x)
	{
		int i = opt % c_x_length + 1;
		int j = opt / c_x_length + 1;

		double x, y;
// D 
		x = (c_h*(i - 1) + c_h*i) / 2.;
		//y = (a_y[j + 1] + a_y[j]) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		result[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		result[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));
			
	}
}

float get_quad_coord(TriangleResult* result, ComputeParameters* p)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	size_t size(0), n(0);
	int gridSize = 256;
	int blockSize =  512;
	double temp(0);
	n = p->get_inner_matrix_size();


	float elapsedTime;

#ifdef FLAGMAN

	// Start record
	hipEventRecord(start, 0);

	hipMemcpyToSymbol(HIP_SYMBOL(c_tau), &p->tau, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_lb), &p->lb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_rb), &p->rb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_bb), &p->bb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_ub), &p->ub, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_n), &n, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(c_x_length), &result->x_length, sizeof(int));
	temp = 1. / (result->x_length + 1);
	hipMemcpyToSymbol(HIP_SYMBOL(c_h), &temp, sizeof(double));

	temp = (1. + p->currentTimeLevel * p->tau) / 10.;
	hipMemcpyToSymbol(HIP_SYMBOL(c_tau_to_current_time_level), &temp, sizeof(double));

	temp = p->b * p->tau;
	hipMemcpyToSymbol(HIP_SYMBOL(c_tau_b), &temp, sizeof(double));

	temp = C_pi_device / 2.;
	hipMemcpyToSymbol(HIP_SYMBOL(c_pi_half), &temp, sizeof(double));

	size = 2 * sizeof(double)*n;
	
	
		int deviceCount = get_cuda_device_count();
	printf("Device Count = %d\n", deviceCount);
	
	 omp_set_num_threads(6);
	 #pragma omp parallel
	 {
	 
	 double *res= NULL;
	 
	 checkCuda(hipMalloc((void**)&(res), size) );
	 int cpuId = omp_get_thread_num();
	 hipSetDevice(cpuId);
	 
	// можно это ядро раскидать на карточки 
	// Вариант 1) На 1 карте считать first1, second1, third1, а на второй считать first2, second2, third2
	// Вариант 2) На 1 карте считать first1, на второй second1 и т. д.
	
	if (cpuId == 0 || cpuId == 1)
	{
	get_square_coord_first1_2 << <gridSize, blockSize >> >(res);
 }
 if (cpuId == 2)
	{
	get_square_coord_second1 << <gridSize, blockSize >> >(res);
 }
 if (cpuId == 3 || cpuId == 4)
 {
 get_square_coord_third1_2 << <gridSize, blockSize >> >(res);
 
 }
 
 if (cpuId == 5)
 {
 get_square_coord_second2 << <gridSize, blockSize >> >(res);
 
 }
	 
	 printf("Hi! I'm %d thread!\n", cpuId);
	 if (cpuId == 0)
	 {
	
	 hipMemcpy(result->first1, res, size, hipMemcpyDeviceToHost); 
	 }
	 else if (cpuId = 1)
	 
	{
	
	 hipMemcpy(result->first2, res, size, hipMemcpyDeviceToHost);	 
	 }
	 else if (cpuId = 2)
	 {
	 
	 hipMemcpy(result->second1, res, size, hipMemcpyDeviceToHost);
	 }
	 else if (cpuId = 3)
	 {
	 
	 hipMemcpy(result->second2, res, size, hipMemcpyDeviceToHost);
	 }
	 else if (cpuId = 4)
	 {
	 
	 hipMemcpy(result->third1, res, size, hipMemcpyDeviceToHost);
	 }
	 else if (cpuId == 5)
	 {
	 hipMemcpy(result->third2, res, size, hipMemcpyDeviceToHost);
	 }
 hipFree(res);
	hipDeviceReset();
	 
	 }
	  hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	
	hipDeviceReset();
	return elapsedTime;
	#else 
	double *first1 = NULL, *second1 = NULL, *third1 = NULL, *first2 = NULL, *second2 = NULL, *third2 = NULL;

	// Start record
	hipEventRecord(start, 0);

	hipMemcpyToSymbol(HIP_SYMBOL(c_tau), &p->tau, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_lb), &p->lb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_rb), &p->rb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_bb), &p->bb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_ub), &p->ub, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_n), &n, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(c_x_length), &result->x_length, sizeof(int));
	temp = 1. / (result->x_length + 1);
	hipMemcpyToSymbol(HIP_SYMBOL(c_h), &temp, sizeof(double));

	temp = (1. + p->currentTimeLevel * p->tau) / 10.;
	hipMemcpyToSymbol(HIP_SYMBOL(c_tau_to_current_time_level), &temp, sizeof(double));

	temp = p->b * p->tau;
	hipMemcpyToSymbol(HIP_SYMBOL(c_tau_b), &temp, sizeof(double));

	temp = C_pi_device / 2.;
	hipMemcpyToSymbol(HIP_SYMBOL(c_pi_half), &temp, sizeof(double));

	size = 2 * sizeof(double)*n;
	checkCuda(hipMalloc((void**)&(first1), size) );
	checkCuda(hipMalloc((void**)&(second1), size));
	checkCuda(hipMalloc((void**)&(third1), size) );
	checkCuda(hipMalloc((void**)&(first2), size) );
	checkCuda(hipMalloc((void**)&(second2), size));
	checkCuda(hipMalloc((void**)&(third2), size) );  

	// можно это ядро раскидать на карточки 
	// Вариант 1) На 1 карте считать first1, second1, third1, а на второй считать first2, second2, third2
	// Вариант 2) На 1 карте считать first1, на второй second1 и т. д.
	get_square_coord << <gridSize, blockSize >> >(first1, second1, third1, first2, second2, third2);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(result->first1, first1, size, hipMemcpyDeviceToHost);
	hipMemcpy(result->second1, second1, size, hipMemcpyDeviceToHost);
	hipMemcpy(result->third1, third1, size, hipMemcpyDeviceToHost);
	hipMemcpy(result->first2, first2, size, hipMemcpyDeviceToHost);
	hipMemcpy(result->second2, second2, size, hipMemcpyDeviceToHost);
	hipMemcpy(result->third2, third2, size, hipMemcpyDeviceToHost);

	hipFree(first1);
	hipFree(second1);
	hipFree(third1);
	hipFree(first2);
	hipFree(second2);
	hipFree(third2);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceReset();
	return elapsedTime;
	
	#endif
}